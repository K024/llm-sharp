#include "hip/hip_runtime.h"
// Copyright (c) OpenMMLab. All rights reserved.

#include "../macro.h"
#include "../kernels/reduce_kernel_utils.cuh"
#include "../utils/cuda_type_utils.cuh"
#include "../utils/cuda_utils.h"

namespace turbomind {

// fp16, bf16
// n is divided by 2 for this impl
template<typename T>
__global__ void rootMeanSquareNorm(T* out, const T* input, const T* scale, float eps, int m, int n)
{
    using T2 = typename TypeConverter<T>::Type;
    __shared__ float s_inv_mean;
    float            mean = 0.f;

    T2*       out_ptr   = (T2*)out;
    const T2* input_ptr = (const T2*)input;
    const T2* scale_ptr = (const T2*)scale;

    for (uint idx = threadIdx.x; idx < n; idx += blockDim.x) {
        float2 tmp2 = cuda_cast<float2>(input_ptr[blockIdx.x * n + idx]);
        mean += tmp2.x * tmp2.x;
        mean += tmp2.y * tmp2.y;
    }

    mean = blockReduceSum<float>(mean);
    if (threadIdx.x == 0) {
        s_inv_mean = rsqrt(.5f * mean / (float)n + eps);
    }
    __syncthreads();

    for (uint idx = threadIdx.x; idx < n; idx += blockDim.x) {
        float2 tmp2                   = cuda_cast<float2>(input_ptr[blockIdx.x * n + idx]);
        float2 sca2                   = cuda_cast<float2>(scale_ptr[idx]);
        tmp2.x                        = tmp2.x * s_inv_mean * sca2.x;
        tmp2.y                        = tmp2.y * s_inv_mean * sca2.y;
        out_ptr[blockIdx.x * n + idx] = cuda_cast<T2>(tmp2);
    }
}

template<>
__global__ void rootMeanSquareNorm(float* out, const float* input, const float* scale, float eps, int m, int n)
{
    __shared__ float s_inv_mean;
    float            mean = 0.f;

    for (uint idx = threadIdx.x; idx < n; idx += blockDim.x) {
        float tmp = input[blockIdx.x * n + idx];
        mean += tmp * tmp;
    }

    mean = blockReduceSum<float>(mean);
    if (threadIdx.x == 0) {
        s_inv_mean = rsqrt(mean / static_cast<float>(n) + eps);
    }
    __syncthreads();

    for (uint idx = threadIdx.x; idx < n; idx += blockDim.x) {
        float tmp                 = input[blockIdx.x * n + idx];
        out[blockIdx.x * n + idx] = tmp * s_inv_mean * scale[idx];
    }
}

template<typename T>
void invokeRootMeanSquareNorm(T* out, const T* input, const T* scale, float eps, int m, int n, hipStream_t stream)
{
    if (sizeof(T) == 2) {
        FT_CHECK(n % 2 == 0);
        n /= 2;
    }
    dim3 grid(m);
    dim3 block(std::min(n, 1024));
    rootMeanSquareNorm<<<grid, block, 0, stream>>>(out, input, scale, eps, m, n);
}

template void invokeRootMeanSquareNorm(float*, const float*, const float*, float, int, int, hipStream_t);
template void invokeRootMeanSquareNorm(half*, const half*, const half*, float, int, int, hipStream_t);

// #ifdef ENABLE_BF16

// template void invokeRootMeanSquareNorm(__hip_bfloat16*, const __hip_bfloat16*, float, int, int, hipStream_t);

// #endif

}  // namespace turbomind
